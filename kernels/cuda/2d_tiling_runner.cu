#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <sys/time.h>
#include <vector>
#include <iostream>
#include "2d_tiling.cuh"

#define CHECK_CUDA(err) { cudaCheck((err), __FILE__, __LINE__); }
#define CHECK_LAST_CUDA_ERROR() { cudaCheck(hipGetLastError(), __FILE__, __LINE__); }

void cudaCheck(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
               hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void randomize_matrix(float *mat, int N) {
    struct timeval time {};
    gettimeofday(&time, nullptr);
    srand(time.tv_usec);
    for (int i = 0; i < N; i++) {
        float tmp = (float)(rand() % 5) + 0.01 * (rand() % 5);
        tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
        mat[i] = tmp;
    }
}

bool verify_matrix(float *matRef, float *matOut, int N) {
    double diff = 0.0;
    int i;
    for (i = 0; i < N; i++) {
        diff = std::fabs(matRef[i] - matOut[i]);
        if (diff > 0.01) {
            printf("Divergence! Should %5.2f, Is %5.2f (Diff %5.2f) at %d\n",
                   matRef[i], matOut[i], diff, i);
            return false;
        }
    }
    return true;
}

void runSgemm2DBlocktiling(int M, int N, int K, float alpha, const float *A,
                         const float *B, float beta, float *C) {
    const uint BM = 128;
    const uint BN = 128;
    const uint BK = 8;
    const uint TM = 8;
    const uint TN = 8;

    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));

    sgemm2DBlocktiling<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
    CHECK_LAST_CUDA_ERROR();
}

int main(int argc, char **argv) {
    std::vector<int> SIZE = {128, 256, 512, 1024, 2048, 4096};
    int max_size = SIZE[SIZE.size() - 1];
    std::cout << "Max size: " << max_size << std::endl;
    
    float alpha = 0.5f;
    float beta = 3.0f;
    
    // Host memory allocation
    float *A = (float *)malloc(sizeof(float) * max_size * max_size);
    float *B = (float *)malloc(sizeof(float) * max_size * max_size);
    float *C = (float *)malloc(sizeof(float) * max_size * max_size);
    float *C_ref = (float *)malloc(sizeof(float) * max_size * max_size);
    if (!A || !B || !C || !C_ref) {
        printf("Host memory allocation failed\n");
        exit(1);
    }

    // Initialize matrices
    randomize_matrix(A, max_size * max_size);
    randomize_matrix(B, max_size * max_size);
    randomize_matrix(C, max_size * max_size);
    
    // Device memory allocation
    float *dA, *dB, *dC, *dC_ref;
    CHECK_CUDA(hipMalloc((void **)&dA, sizeof(float) * max_size * max_size));
    CHECK_CUDA(hipMalloc((void **)&dB, sizeof(float) * max_size * max_size));
    CHECK_CUDA(hipMalloc((void **)&dC, sizeof(float) * max_size * max_size));
    CHECK_CUDA(hipMalloc((void **)&dC_ref, sizeof(float) * max_size * max_size));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasStatus_t cublas_status = hipblasCreate(&handle);
    if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS initialization failed\n");
        exit(1);
    }

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    int repeat_times = 50;
    
    for (int size : SIZE) {
        int M = size;
        int N = size;
        int K = size;
        
        std::cout << "dimensions(m=n=k) " << M << ", alpha: " << alpha
                  << ", beta: " << beta << std::endl;
        
        // Copy current size data to device
        CHECK_CUDA(hipMemcpy(dA, A, sizeof(float) * M * K, hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(dB, B, sizeof(float) * K * N, hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(dC, C, sizeof(float) * M * N, hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(dC_ref, C, sizeof(float) * M * N, hipMemcpyHostToDevice));
        
        // Run reference cuBLAS
        cublas_status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                   N, M, K,
                                   &alpha,
                                   dB, N,
                                   dA, K,
                                   &beta,
                                   dC_ref, N);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            printf("cuBLAS GEMM failed\n");
            exit(1);
        }
        CHECK_CUDA(hipDeviceSynchronize());
                    
        // Run our implementation once for correctness
        runSgemm2DBlocktiling(M, N, K, alpha, dA, dB, beta, dC);
        CHECK_CUDA(hipDeviceSynchronize());
        
        // Copy results back for verification
        CHECK_CUDA(hipMemcpy(C, dC, sizeof(float) * M * N, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(C_ref, dC_ref, sizeof(float) * M * N, hipMemcpyDeviceToHost));
        
        if (!verify_matrix(C_ref, C, M * N)) {
            printf("Verification failed for size %d!\n", size);
            exit(1);
        }
        
        // Timing runs
        CHECK_CUDA(hipEventRecord(start));
        for (int j = 0; j < repeat_times; j++) {
            runSgemm2DBlocktiling(M, N, K, alpha, dA, dB, beta, dC);
        }
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));
        
        float milliseconds = 0;
        CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
        float seconds = milliseconds / 1000.0f;
        
        // Use double for higher precision in GFLOPS calculation
        double flops = (double)2.0 * M * N * K;
        double gflops = (repeat_times * flops * 1e-9) / seconds;
        
        printf("Average elapsed time: (%7.6f) s, performance: (%7.1f) GFLOPS. size: (%d).\n",
               seconds / repeat_times,
               gflops, M);
    }

    // Cleanup
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipFree(dA));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipFree(dC));
    CHECK_CUDA(hipFree(dC_ref));
    hipblasDestroy(handle);
    free(A);
    free(B);
    free(C);
    free(C_ref);

    return 0;
}
